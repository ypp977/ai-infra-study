#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void mmul_native(const float* A, const float* B, float* C, int N)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < N && col < N)
    {
        float sum = 0;
        for (int k = 0; k < N; k++)
        {
            sum += A[row * N + k] * B[k * N + col];
        }
        C[row * N + col] = sum;
    }
}

int main()
{
    const int N = 512;
    size_t bytes = N * N * sizeof(float);

    float host_a[N * N], host_b[N * N], host_c[N * N];

    for (int i = 0; i < N * N; i++)
    {
        host_a[i] = 1.0f;
        host_b[i] = 2.0f;
    }

    float *device_a, *device_b, *device_c;
    hipMalloc(&device_a, bytes);
    hipMalloc(&device_b, bytes);
    hipMalloc(&device_c, bytes);

    hipMemcpy(device_a, host_a, bytes, hipMemcpyHostToDevice);
    hipMemcpy(device_b, host_b, bytes, hipMemcpyHostToDevice);

    dim3 block(16, 16);
    dim3 grid((N + block.x - 1) / block.x, (N + block.y - 1) / block.y);

    float ms = 0;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    mmul_native<<<grid, block>>>(device_a, device_b, device_c, N);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&ms, start, stop);
    hipMemcpy(host_c, device_c, bytes, hipMemcpyDeviceToHost);

    printf("Naive GEMM time = %.3f ms, result C[0]=%.1f\n", ms, host_c[0]);

    hipFree(device_a);
    hipFree(device_b);
    hipFree(device_c);

    return 0;
}
