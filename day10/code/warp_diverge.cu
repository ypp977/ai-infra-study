#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void kernel_if(int* out, const int* in, int N)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < N)
    {
        if (in[tid] % 2 == 0)
        {
            out[tid] = in[tid] * 2;
        }
        else
        {
            out[tid] = in[tid] * 3;
        }
    }
}

__global__ void kernel_selp(int* out, const int* in, int N)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < N)
    {
        int val = in[tid];
        // 三目运算避免分支发散
        out[tid] = (val % 2 == 0) ? (val * 2) : (val * 3);
    }
}

int main()
{
    const int N = 1 << 20;
    size_t bytes = N * sizeof(int);

    int* host_in = (int*)malloc(bytes);
    int* host_out = (int*)malloc(bytes);
    for (int i = 0; i < N; i++)
    {
        host_in[i] = i;
    }

    int *device_in, *device_out;
    hipMalloc(&device_in, bytes);
    hipMalloc(&device_out, bytes);
    hipMemcpy(device_in, host_in, bytes, hipMemcpyHostToDevice);

    dim3 block(256);
    dim3 grid((N + block.x - 1) / block.x);

    // 计时
    hipEvent_t start, stop;
    float ms;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    kernel_if<<<grid, block>>>(device_out, device_in, N);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ms, start, stop);
    printf("if-else kernel: %.3f ms\n", ms);

    hipEventRecord(start);
    kernel_selp<<<grid, block>>>(device_out, device_in, N);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ms, start, stop);
    printf("selp kernel: %.3f ms\n", ms);

    hipFree(device_in);
    hipFree(device_out);

    free(host_in);
    free(host_out);

    return 0;
}
