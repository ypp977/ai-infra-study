#include <hip/hip_runtime.h>
#include <stdio.h>

// 简单的计算kernel (模拟耗时计算)
__global__ void computer(float* data, int N)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N)
    {
        float x = data[i];
        for (int j = 0; j < 10000; j++)
        {
            x = x * 0.999f + 0.001f;
        }
        data[i] = x;
    }
}

int main()
{
    const int N = 1 << 24;
    size_t bytes = N * sizeof(float);

    float *host_data, *device_data;
    hipHostMalloc(&host_data, bytes, hipHostMallocDefault); // 页锁定内存(必须)
    hipMalloc(&device_data, bytes);

    for (int i = 0; i < N; i++)
    {
        host_data[i] = 1.0f;
    }

    dim3 block(256);
    dim3 grid((N + block.x - 1) / block.x);

    // 同步版本
    hipEvent_t start, stop;
    float ms_sync, ms_async;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    // 同步拷贝 H2D
    hipMemcpy(device_data, host_data, bytes, hipMemcpyHostToDevice);
    // 计算
    computer<<<grid, block>>>(device_data, N);
    // 同步拷贝 D2H
    hipMemcpy(host_data, device_data, bytes, hipMemcpyDeviceToHost);

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&ms_sync, start, stop);

    // 异步版本
    hipStream_t s1, s2;
    hipStreamCreate(&s1);
    hipStreamCreate(&s2);

    hipEventRecord(start);
    // H2D 异步拷贝
    hipMemcpyAsync(device_data, host_data, bytes, hipMemcpyHostToDevice, s1);
    // 计算放到另一个stream
    computer<<<grid, block, 0, s2>>>(device_data, N);
    // D2H 异步拷贝
    hipMemcpyAsync(host_data, device_data, bytes, hipMemcpyDeviceToHost, s1);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ms_async, start, stop);

    printf("Sync  version time: %.3f ms\n", ms_sync);
    printf("Async version time: %.3f ms\n", ms_async);

    hipStreamDestroy(s1);
    hipStreamDestroy(s2);
    hipFree(device_data);
    hipHostFree(host_data);

    return 0;
}
