#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void touch(float* data, long N)
{
    long i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N)
    {
        data[i] += 1.0f;
    }
}

int main(int argc, char* argv[])
{
    // N 太大可能导致系统直接OOM
    long N = (long)1e9; // 默认 1e9 (~4 GB)
    if (argc > 1)
    {
        N = atol(argv[1]); // 可以从命令行传 N
    }
    size_t bytes = N * sizeof(float);

    printf("Allocating %.2f GB Unified Memory...\n", bytes / 1e9);

    float* data;
    hipMallocManaged(&data, bytes); // unified memory

    dim3 block(256);
    dim3 grid((N + block.x - 1) / block.x);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    touch<<<grid, block>>>(data, N);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float ms;
    hipEventElapsedTime(&ms, start, stop);

    printf("Kernel with UM (N=%ld, %.2f GB): %.3f ms\n", N, bytes / 1e9, ms);

    hipFree(data);

    return 0;
}
