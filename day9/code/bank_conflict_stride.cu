#include <hip/hip_runtime.h>
#include <stdio.h>

// 冲突版本： stride = 17
__global__ void conflict(float* out)
{
    __shared__ float shared_data[32 * 17];
    int tid = threadIdx.x;
    // 多个线程映射到同一个bank
    shared_data[tid * 17] = tid;
    __syncthreads();
    out[tid] = shared_data[tid * 17];
}

// 无冲突版本： stride = 17 + padding
__global__ void no_conflict(float* out)
{
    __shared__ float shared_data[32 * 17 + 1]; // padding +1
    int tid = threadIdx.x;
    // padding 打散 bank 映射
    shared_data[tid * 17] = tid;
    __syncthreads();
    out[tid] = shared_data[tid * 17];
}

// 计时封装函数
float run_and_time(void (*kernel)(float*), float* device_out, int N)
{
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    kernel<<<1, N>>>(device_out);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float ms = 0;
    hipEventElapsedTime(&ms, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return ms;
}

int main()
{
    // warp 内 32 线程
    const int N = 32;
    size_t bytes = N * sizeof(float);

    float host_out[N];
    float* device_out;
    hipMalloc(&device_out, bytes);

    // 计时并运行冲突版本
    float time1 = run_and_time(conflict, device_out, N);
    hipMemcpy(host_out, device_out, bytes, hipMemcpyDeviceToHost);
    printf("Conflict kernel (%.6f ms)\n", time1);
    for (int i = 0; i < 5; i++)
    {
        printf("out[%d] = %.1f ", i, host_out[i]);
    }
    printf("\n");

    // 计时并运行无冲突版本
    float time2 = run_and_time(no_conflict, device_out, N);
    hipMemcpy(host_out, device_out, bytes, hipMemcpyDeviceToHost);
    printf("NO Conflict kernel (%.6f ms)\n", time2);
    for (int i = 0; i < 5; i++)
    {
        printf("out[%d] = %.1f ", i, host_out[i]);
    }
    printf("\n");

    hipFree(device_out);
    return 0;
}
