#include <hip/hip_runtime.h>
#include <stdio.h>

// 常量内存
#define COEF_SIZE 1024
__constant__ float device_coef[COEF_SIZE]; // GPU常量内存

__global__ void kernel_const(const float* in, float* out, int N)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N)
    {
        float val = in[i];
        for (int j = 0; j < 1000; j++)
        {
            val *= device_coef[j % COEF_SIZE];
        }
        out[i] = val;
    }
}

__global__ void kernel_global(const float* in, float* out, const float* coef, int N)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N)
    {
        float val = in[i];
        for (int j = 0; j < 1000; j++)
        {
            val *= coef[j % COEF_SIZE];
        }
        out[i] = val;
    }
}

int main()
{
    const int N = 1 << 24; // 16M 元素
    size_t bytes = N * sizeof(float);

    float* host_in = (float*)malloc(bytes);
    float* host_out = (float*)malloc(bytes);
    for (int i = 0; i < N; i++)
    {
        host_in[i] = 1.0f;
    }

    float *device_in, *device_out, *device_coef_global;
    hipMalloc(&device_in, bytes);
    hipMalloc(&device_out, bytes);
    hipMalloc(&device_coef_global, COEF_SIZE * sizeof(float));

    hipMemcpy(device_in, host_in, bytes, hipMemcpyHostToDevice);

    float host_coef[COEF_SIZE];
    for (int i = 0; i < COEF_SIZE; i++)
    {
        host_coef[i] = 1.0f;
    }
    // 把 coef 放到 constant memory
    hipMemcpyToSymbol(HIP_SYMBOL(device_coef), host_coef, COEF_SIZE * sizeof(float));
    // 把 coef 放到 global memory
    hipMemcpy(device_coef_global, host_coef, COEF_SIZE * sizeof(float), hipMemcpyHostToDevice);

    dim3 block(256);
    dim3 grid((N + block.x - 1) / block.x);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    float ms;

    // constant memory
    hipEventRecord(start);
    kernel_const<<<grid, block>>>(device_in, device_out, N);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ms, start, stop);
    printf("Const memory kernel: %.3f ms\n", ms);

    // global memory
    hipEventRecord(start);
    kernel_global<<<grid, block>>>(device_in, device_out, device_coef_global, N);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ms, start, stop);
    printf("Global memory kernel: %.3f ms\n", ms);

    hipFree(device_in);
    hipFree(device_out);
    hipFree(device_coef_global);

    free(host_in);
    free(host_out);

    return 0;
}
