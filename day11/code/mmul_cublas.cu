#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

int main()
{
    int N = 1024;
    size_t size = N * N * sizeof(float);

    float* host_a = (float*)malloc(size);
    float* host_b = (float*)malloc(size);
    float* host_c = (float*)malloc(size);

    for (int i = 0; i < N * N; i++)
    {
        host_a[i] = 1.0f;
        host_b[i] = 1.0f;
    }

    float *device_a, *device_b, *device_c;
    hipMalloc((void**)&device_a, size);
    hipMalloc((void**)&device_b, size);
    hipMalloc((void**)&device_c, size);

    hipMemcpy(device_a, host_a, size, hipMemcpyHostToDevice);
    hipMemcpy(device_b, host_b, size, hipMemcpyHostToDevice);

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    float alpha = 1.0f, beta = 0.0f;

    int repeat = 5; // 统计 5 次
    float total_ms = 0.0f;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    // 排除第一次调用overhead
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha, device_b, N, device_a, N,
                   &beta, device_c, N);
    hipDeviceSynchronize();

    for (int i = 0; i < repeat; i++)
    {
        hipEventRecord(start);
        hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha, device_b, N, device_a, N,
                       &beta, device_c, N);

        hipEventRecord(stop);
        hipEventSynchronize(stop);
        float ms;

        hipEventElapsedTime(&ms, start, stop);
        total_ms += ms;
    }
    float avg_ms = total_ms / repeat;
    hipMemcpy(host_c, device_c, size, hipMemcpyDeviceToHost);

    printf("cuBLAS N=%d, Time: %f ms\n", N, avg_ms);
    printf("host_c[0] = %f\n", host_c[0]);

    hipblasDestroy(handle);

    hipFree(device_a);
    hipFree(device_b);
    hipFree(device_c);

    free(host_a);
    free(host_b);
    free(host_c);

    return 0;
}
