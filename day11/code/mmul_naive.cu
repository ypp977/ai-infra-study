
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void mmul_native(const float* A, const float* B, float* C, int N)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < N && col < N)
    {
        float sum = 0.0f;
        for (int k = 0; k < N; k++)
        {
            sum += A[row * N + k] * B[k * N + col];
        }
        C[row * N + col] = sum;
    }
}

int main()
{
    int N = 1024;
    size_t size = N * N * sizeof(float);

    float *host_a, *host_b, *host_c, *device_a, *device_b, *device_c;
    host_a = (float*)malloc(size);
    host_b = (float*)malloc(size);
    host_c = (float*)malloc(size);
    for (int i = 0; i < N * N; i++)
    {
        host_a[i] = 1.0f;
        host_b[i] = 1.0f;
    }

    hipMalloc(&device_a, size);
    hipMalloc(&device_b, size);
    hipMalloc(&device_c, size);

    hipMemcpy(device_a, host_a, size, hipMemcpyHostToDevice);
    hipMemcpy(device_b, host_b, size, hipMemcpyHostToDevice);

    dim3 block(16, 16);
    dim3 grid((N + block.x - 1) / block.x, (N + block.y - 1) / block.y);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    mmul_native<<<grid, block>>>(device_a, device_b, device_c, N);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float ms;
    hipEventElapsedTime(&ms, start, stop);

    hipMemcpy(host_c, device_c, size, hipMemcpyDeviceToHost);

    printf("GPU naive done, host_c[0]=%.1f , time = %.4f\n", host_c[0], ms);

    hipFree(device_a);
    hipFree(device_b);
    hipFree(device_c);

    free(host_a);
    free(host_b);
    free(host_c);

    return 0;
}
